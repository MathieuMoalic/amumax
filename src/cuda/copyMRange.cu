
#include <hip/hip_runtime.h>
extern "C" __global__ void
copyMRange(float* __restrict__ dst,
          float* __restrict__ src,
          int Nx, int Ny, int Nz,          // full volume dims
          int dx0, int dy0, int dz0,       // dst origin
          int sx0, int sy0, int sz0,       // src origin
          int W, int H, int D,             // box size
          int wrap)                        // 0=clip, 1=wrap
{
    int x = blockIdx.x * blockDim.x + threadIdx.x; // [0..W)
    int y = blockIdx.y * blockDim.y + threadIdx.y; // [0..H)
    int z = blockIdx.z * blockDim.z + threadIdx.z; // [0..D)
    if (x >= W || y >= H || z >= D) return;

    int sx = sx0 + x, sy = sy0 + y, sz = sz0 + z;
    int dx = dx0 + x, dy = dy0 + y, dz = dz0 + z;

    if (wrap) {
        // positive modulo
        sx = (sx % Nx + Nx) % Nx;
        sy = (sy % Ny + Ny) % Ny;
        sz = (sz % Nz + Nz) % Nz;
        dx = (dx % Nx + Nx) % Nx;
        dy = (dy % Ny + Ny) % Ny;
        dz = (dz % Nz + Nz) % Nz;
    } else {
        if (sx < 0 || sy < 0 || sz < 0 || sx >= Nx || sy >= Ny || sz >= Nz) return;
        if (dx < 0 || dy < 0 || dz < 0 || dx >= Nx || dy >= Ny || dz >= Nz) return;
    }

    size_t sidx = ((size_t)sz * Ny + sy) * Nx + sx;
    size_t didx = ((size_t)dz * Ny + dy) * Nx + dx;

    dst[didx] = src[sidx];
}
